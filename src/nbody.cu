#include "hip/hip_runtime.h"
//#include "jbutil.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <algorithm>
#include <iostream>
#include <fstream>
#include <string>
#include <sstream>
#include <vector>
#include <limits>
#include <math.h>
#include <random>

//#include "vec2.h"
#include "vec2f.h"
#include "jbutil.h"
#include "nbodyConfig.h"

#define CUDA_SYNC_CHECK()                                                      \
    do                                                                         \
    {                                                                          \
        hipDeviceSynchronize();                                               \
        hipError_t error = hipGetLastError();                                \
        if( error != hipSuccess )                                             \
        {                                                                      \
            std::stringstream ss;                                              \
            ss << "CUDA error on synchronize with error '"                     \
               << hipGetErrorString( error )                                  \
               << "' (" __FILE__ << ":" << __LINE__ << ")\n";                  \
            throw std::runtime_error( ss.str().c_str() );                        \
        }                                                                      \
    } while( 0 )

#define MAX_RUNS 1
#define THREADS_PER_BLOCK 128
#define GRAV_CONSTANT 6.67408e-11f

/*
 * Definitions for field dimensions
 */
int fieldWidth;
int doubleFieldWidth;// = fieldWidth << 1;
int fieldHeight;// = 2000;
int doubleFieldHeight;// = fieldHeight << 1;

struct BodiesData
{
	int numBodies;
	size_t size;
	void* contiguousData;
	Vec2f* Positions;
	Vec2f* Velocities;
	float* Masses;
	float* Radii;

	void* d_contiguousData;

	BodiesData() : numBodies(), size(), contiguousData(), Positions(), Velocities(), Masses(), Radii(), d_contiguousData()
	{
	}

	void alloc(int bodies)
	{
		numBodies = bodies;
		size = numBodies * (sizeof(Vec2f) + sizeof(Vec2f) + sizeof(float) + sizeof(float));
		contiguousData = (void*)malloc(size);
		if(contiguousData == NULL)
		{
			printf("Failed to allocate body data");
			exit(0); //fail since program can't continue
		}
		//Splitting the contiguous data between the required arrays
		Positions = (Vec2f*) contiguousData;
		//printf("cont: %p\nPositions: %p\n", &contiguousData, );
		Velocities = (Vec2f*) &Positions[numBodies];
		Masses = (float*) &Velocities[numBodies];
		Radii = (float* ) &Masses[numBodies];
		d_contiguousData = nullptr;
	}

	void freeData()
	{
		free(contiguousData);
		hipFree(d_contiguousData);
		d_contiguousData = nullptr;
	}

	void uploadToDevice(hipStream_t stream = 0)
	{
		//prevent reallocating memory
		if(d_contiguousData == nullptr)
		{
			hipMalloc((void**)&d_contiguousData, size);
//			hipMemcpyAsync(d_contiguousData, contiguousData, size, hipMemcpyHostToDevice, stream);
			hipMemcpy(d_contiguousData, contiguousData, size, hipMemcpyHostToDevice);
		}
	}

	//Make sure to delete previous data before calling this
	BodiesData& operator= (const BodiesData& newData)
	{
		contiguousData = newData.contiguousData;
		numBodies = newData.numBodies;
		size = newData.size;
		Positions = newData.Positions;
		Velocities = newData.Velocities;
		Masses = newData.Masses;
		Radii = newData.Radii;
		d_contiguousData = newData.d_contiguousData;
		return *this;
	}

	void printData()
	{
		for(int i = 0 ; i < numBodies ; ++i)
		{
			printf("Body #%d\n", i);
			printf("Position: (%.4f, %.4f)\n", Positions[i].X, Positions[i].Y);
			printf("Velocity: (%.4f, %.4f)\n", Velocities[i].X, Velocities[i].Y);
			printf("Mass: %.4f\n", Masses[i]);
			printf("Radius: %.4f\n", Radii[i]);
			printf("--------------\n");
		}
	}
};

//typedef struct rgb_data {
//	int r;
//	int g;
//	int b;
//} RGB;

//Device global variables to prevent copying to host every iterationd
//__device__ Particle* deviceBodies;
//__device__ float* deviceUpdatedMasses;
//__device__ int deviceNumBodies;

__device__ inline bool areParticlesColliding(const Vec2f& p0, const float r0, const Vec2f& p1, const float r1)
{
	Vec2f direction = p1 - p0;
	float distance = (direction.X * direction.X) + (direction.Y * direction.Y);
	return distance <= (r0 + r1) * (r0 + r1);
}

/*
 * Compute forces of particles exerted on one another
 */
//Particle* d_bodies, float* updatedMasses, Vec2f* updatedVelocities, float* updatedRadii,
__global__ void /*__launch_bounds__(THREADS_PER_BLOCK, 2)*/ ComputeForces(void* bodyData, float* updatedMasses, Vec2f* updatedVelocities,
		float* updatedRadii, int bodiesNum, float timestep, int fieldWidth, int fieldHeight, int numBlocks)
{
	//Particle* bodies = d_bodies;//deviceBodies;
	int globalThreadIdx = blockIdx.x * blockDim.x + threadIdx.x;
	if(globalThreadIdx < bodiesNum)
	{
		//Subdividing the contiguous array
		int numBodies = bodiesNum;
		Vec2f* positions = (Vec2f*) bodyData;
		Vec2f* velocities = (Vec2f*) &positions[numBodies];
		float* masses = (float*) &velocities[numBodies];
		float* radii = (float*) &masses[numBodies];

		//float* updated_masses = updatedMasses;
		Vec2f direction, force, acceleration;
		force = 0.f, acceleration = 0.f;
		float distance;

		extern __shared__ Vec2f sharedMem[];
		Vec2f* shrdPositions = (Vec2f* )&sharedMem;
		float* shrdMasses = (float*) &shrdPositions[THREADS_PER_BLOCK];
		float* shrdRadii = (float*) &shrdMasses[THREADS_PER_BLOCK];

		Vec2f* blockThreadsPositions =(Vec2f*) &shrdRadii[THREADS_PER_BLOCK];
		float* blockThreadsMasses = (float*) &blockThreadsPositions[THREADS_PER_BLOCK];
		float* blockThreadsRadii = (float*) &blockThreadsMasses[THREADS_PER_BLOCK];
		Vec2f* blockThreadsVelocities = (Vec2f *) &blockThreadsRadii[THREADS_PER_BLOCK];

		Vec2f* shrdUpdatedVelocities = (Vec2f* ) &blockThreadsVelocities[THREADS_PER_BLOCK];

		//Particle &p1 = bodies[j];
		//Loading data for the body that the current thread is handling
	//	const Vec2f& threadBodyPosition = positions[globalThreadIdx];
	//	const float threadBodyMass = masses[globalThreadIdx] ;
	//	const float threadBodyRadius = radii[globalThreadIdx];

		blockThreadsPositions[threadIdx.x] = positions[globalThreadIdx];
		blockThreadsMasses[threadIdx.x] = masses[globalThreadIdx] ;
		blockThreadsRadii[threadIdx.x] = radii[globalThreadIdx];
		blockThreadsVelocities[threadIdx.x] = velocities[globalThreadIdx];

		float updatedBodyMass = blockThreadsMasses[threadIdx.x];//threadBodyMass;
		float updatedBodyRadius = blockThreadsRadii[threadIdx.x];
		int globalBodyIdx;
		int shIdx;
		bool skip = true;
		int innerLoopLimit;
		bool deleted = false;
		for (int k = 0; k < numBlocks; ++k)
		{
			//Loading the next p bodies (p = threads per block)
			globalBodyIdx = (globalThreadIdx + (THREADS_PER_BLOCK * k)) % numBodies;
			//if(blockIdx.x == 0) printf("threadIdx: %d\nglobalId: %d\n\n", threadIdx.x, globalBodyIdx);
			shrdPositions[threadIdx.x] = positions[globalBodyIdx];
			shrdMasses[threadIdx.x] = masses[globalBodyIdx];
			shrdRadii[threadIdx.x] = radii[globalBodyIdx];
			__syncthreads();

			//printf("[%d] Mass: %.3f\nPosition: (%.3f, %.3f)\nRadius: %.3f\n",
			//		globalThreadIdx, blockThreadsMasses[threadIdx.x], blockThreadsPositions[threadIdx.x].X, blockThreadsPositions[threadIdx.x].Y, blockThreadsRadii[threadIdx.x]);

			//if this is the last block and less bodies then max threads are present, shared mem access needs to be limited
			innerLoopLimit = k == numBlocks - 1 ? numBodies % (THREADS_PER_BLOCK + 1) : THREADS_PER_BLOCK;
			for (int shrdOffset = 0; shrdOffset < innerLoopLimit ; ++shrdOffset)
			{
				//If globalBodyIdx and globalThreadIdx are equal, we want to ensure that bodies are still compared in shared mem, hence the skip
				if (skip && globalBodyIdx == globalThreadIdx)
				{
					skip = false;
					continue;
				}

				//Ensuring threads loop and read every shared location concurrently
				shIdx = (threadIdx.x + shrdOffset) % innerLoopLimit;

				bool intersect = areParticlesColliding(blockThreadsPositions[threadIdx.x], blockThreadsRadii[threadIdx.x],
									shrdPositions[shIdx], shrdRadii[shIdx]);

				//Testing for body - body collision
				if (intersect && (blockThreadsMasses[threadIdx.x] >= shrdMasses[shIdx]))
				{
					//printf("INTERSECTION TYPE 1: [%d] with [%d]\n", globalThreadIdx, shIdx);
					//printf("[%d] Previous radius: %.3f (shared: %.3f)\n", globalThreadIdx, updatedBodyRadius, shrdRadii[shIdx]);
					updatedBodyMass += shrdMasses[shIdx];
					updatedBodyRadius += shrdRadii[shIdx] * 0.1f;
					//printf("[%d] Updated radius: %.3f\n", globalThreadIdx, updatedBodyRadius);
	//				shrdUpdatedMasses[threadIdx.x] = threadBodyMass + shrdMasses[shIdx];
	//				shrdUpdatedRadii[threadIdx.x] = threadBodyRadius + shrdRadii[shIdx];;//p1.Radius += p2.Radius;
					continue;
				}
				else if ( intersect && (blockThreadsMasses[threadIdx.x] < shrdMasses[shIdx]))
				{
					//shrdUpdatedMasses[threadIdx.x] = 0.f;
					//updatedBodyMass = 0.f;
					//printf("INTERSECTION TYPE 2: [%d] with [%d]\n", globalThreadIdx, shIdx);
					deleted = true;
					continue;
				}

				// Compute direction vector
				direction = shrdPositions[shIdx] - blockThreadsPositions[threadIdx.x];
				//printf("Direction: (%.4f, %.4f)\n", direction.X, direction.Y);
				distance = direction.length();
				//printf("Distance [%d] to [%d]: %.3f\n",  globalThreadIdx, shIdx, distance);
				//printf("[%d] to [%d] Radius 1: %.3f\n", globalThreadIdx, shIdx, blockThreadsRadii[threadIdx.x]);
				//printf("[%d] to [%d] Radius 2: %.3f\n", globalThreadIdx, shIdx, shrdRadii[shIdx]);

	#ifndef NDEBUG
				assert(distance != 0);
	#endif
				// Accumulate force
				//Vec2f temp =
				force += (direction * shrdMasses[shIdx]) / (distance * distance * distance);//(direction * shrdMasses[shIdx]) / (distance * distance * distance);
				//printf("[%d] to [%d]\ndirection: (%.3f,%.3f)\ndistance: %.3f\nforce: (%.3f, %.3f)\n shrdMass: %.3f\nUpdatedForce: (%.3f, %.3f)\n",
				//		globalThreadIdx, shIdx, direction.X, direction.Y, distance, temp.X, temp.Y, shrdMasses[shIdx], force.X, force.Y
				//		);
			}
			__syncthreads();
		};
		__syncthreads();
		updatedMasses[globalThreadIdx] =  deleted ? 0 : updatedBodyMass;//shrdUpdatedMasses[threadIdx.x];
		updatedRadii[globalThreadIdx] = updatedBodyRadius;//shrdUpdatedRadii[threadIdx.x];

		//printf("[%d] Final updatedMass: %.3f\n", globalThreadIdx, updatedMasses[globalThreadIdx]);

		// Compute acceleration for body
		acceleration = force * GRAV_CONSTANT;
		//printf("[%d] acceleration: (%.6f, %.6f)\n", globalThreadIdx, acceleration.X, acceleration.Y);
		shrdUpdatedVelocities[threadIdx.x] = acceleration * timestep;

		//Border collision
		if (blockThreadsPositions[threadIdx.x].X + (acceleration.X * timestep) > fieldWidth - blockThreadsRadii[threadIdx.x]
				|| blockThreadsPositions[threadIdx.x].X + (acceleration.X * timestep) < -fieldWidth + blockThreadsRadii[threadIdx.x])
			blockThreadsVelocities[threadIdx.x].X *= -1;
			//shrdUpdatedVelocities[threadIdx.x].X *= -1;
			//updatedVelocities[globalThreadIdx].X *= -1;
		if (blockThreadsPositions[threadIdx.x].Y + (acceleration.Y * timestep) > fieldHeight - blockThreadsRadii[threadIdx.x]
				|| blockThreadsPositions[threadIdx.x].Y + (acceleration.Y * timestep) < -fieldHeight + blockThreadsRadii[threadIdx.x])
			blockThreadsVelocities[threadIdx.x].Y *= -1;
			//shrdUpdatedVelocities[threadIdx.x].Y *= -1;
			//updatedVelocities[globalThreadIdx].Y *= -1;

		//printf("[%d] updatedMass: %.3f\nupdatedRadius: %.3f\nVel: (%.3f, %.3f)\nupdatedVel: (%.3f, %.3f)\n",
		//		globalThreadIdx, updatedBodyMass, updatedBodyRadius,
		//		blockThreadsVelocities[threadIdx.x].X, blockThreadsVelocities[threadIdx.x].Y,
		//		shrdUpdatedVelocities[threadIdx.x].X, shrdUpdatedVelocities[threadIdx.x].Y);
		velocities[globalThreadIdx] = blockThreadsVelocities[threadIdx.x] + shrdUpdatedVelocities[threadIdx.x];
	}
	/*printf("Mass (p%d) : %.5f\n", (int)j, p1.Mass);
	 printf("Mass (p%d) : %.5f\n", (int)k, p2.Mass);
	 printf("Direction (p%d to p%d) : %.2f, %.2f\n", (int)j, (int)k,direction.X, direction.Y);
	 printf("Distance  (p%d to p%d): %.4f\n", (int)j, (int)k, distance);
	 printf("Force (p%d)  (running sum): %.2f, %.2f\n", (int)j, force.X, force.Y);*/
}

/*
 * Update particle positions
 */

__global__ void MoveBodies(void* bodyData, float* updatedMasses, Vec2f* updatedVelocities,float* updatedRadii,
		int numBodies, float p_deltaT)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	if ( j < numBodies)
	{
		Vec2f* positions = (Vec2f*) bodyData;
		Vec2f* velocities = (Vec2f*) &positions[numBodies];
		float* masses = (float*) &velocities[numBodies];
		float* radii = (float*) &masses[numBodies];
		//Particle* p_bodies = bodies;//deviceBodies;//*bodiesAddr;
		float* updated_masses = updatedMasses;//deviceUpdatedMasses;
//		if (updated_masses[j] != 0.f) {
			//printf("UPDATED MASS [%d]: %.2f\n", (int)j, updated_masses[j]);
		positions[j] += velocities[j] * p_deltaT;//p_bodies[j].Velocity * p_deltaT;
			//velocities[j] = updatedVelocities[j];
		masses[j] = updated_masses[j];
		radii[j] = updatedRadii[j];
//		}
	}
}

__global__ void generateImage(void* bodyData, int numBodies, char* imgData, int width,
		int height, int fieldWidth, int fieldHeight)
{
	//Particle* bodies = d_bodies;
	Vec2f* positions = (Vec2f*) bodyData;
	Vec2f* velocities = (Vec2f*) &positions[numBodies];
	float* masses = (float*) &velocities[numBodies];
	float* radii = (float*) &masses[numBodies];
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	extern __shared__ Vec2f sharedMem[];
	Vec2f* shrdPositions = (Vec2f* )&sharedMem;
	float* shrdRadii = (float*) &shrdPositions[THREADS_PER_BLOCK];

	shrdPositions[threadIdx.x] = positions[i];
	shrdRadii[threadIdx.x] = (radii[i] * width)/fieldWidth;

	//printf("hi\n");

	const int img_width = width;
	const int img_height = height;
	const int doubleFieldWidth = fieldWidth << 1;
	const int doubleFieldHeight = fieldHeight << 1;

	//Since positions can be negative, we add the field width and height to determine the new position in the image ( no negative coords)
	int x_centre_pixel = (int) (((shrdPositions[threadIdx.x].X + fieldWidth) / doubleFieldWidth) * img_width);
	int y_centre_pixel = (int) (((shrdPositions[threadIdx.x].Y + fieldHeight) / doubleFieldHeight) * img_height);

	int x_sq;
	int y_sq;
	int y_min = y_centre_pixel - shrdRadii[threadIdx.x] < 0 ?	0 : y_centre_pixel - shrdRadii[threadIdx.x];
	int y_max = y_centre_pixel + shrdRadii[threadIdx.x] >= img_height ? img_height : y_centre_pixel + shrdRadii[threadIdx.x];
	int x_min =	x_centre_pixel - shrdRadii[threadIdx.x] < 0 ? 0 : x_centre_pixel - shrdRadii[threadIdx.x];
	int x_max = x_centre_pixel + shrdRadii[threadIdx.x] > img_width ?	img_width : x_centre_pixel + shrdRadii[threadIdx.x];

	for (int y = y_min; y < y_max; ++y)
	{
		if (y < 0) y = 0;
		if (y > img_height) y = img_height;
		for (int x = x_min; x < x_max; ++x)
		{
			if (x < 0) x = 0;
			if (x > img_width) x = img_width;
			x_sq = (x - x_centre_pixel) * (x - x_centre_pixel);
			y_sq = (y - y_centre_pixel) * (y - y_centre_pixel);
			if (x_sq + y_sq <= (int)(shrdRadii[threadIdx.x]* shrdRadii[threadIdx.x]))
			{
#ifndef NDEBUG
				assert((x >= 0) && (x <= img_width));
				assert((y >= 0) && (y <= img_height));
#endif
				imgData[(img_width * y) + x] = (char) 0;//static_cast<char>(0);
			};
		}
	}
}

void saveImageToDisk(const std::string &filename, char* imgData, int imgWidth,
		int imgHeight) {
	size_t imageSize = imgWidth * imgHeight;
	std::ofstream outImg;

	outImg.open(filename, std::ofstream::out);
	std::cout << "Saving (" << imgWidth << "x" << imgHeight << ") to disk"
			<< std::endl;
	if (outImg.is_open()) {
		outImg << "P5\n" << imgWidth << " " << imgHeight << "\n255\n";
		for (int i = 0; i < imageSize; ++i) {
			//printf("%d\n", imgData[i]);
			outImg << imgData[i];
		}
		outImg.close();
	}
	else
	{
		std::cerr << "Error writing image to file:" << filename << std::endl
				<< "Ensure the the folder exists" << std::endl;
		exit(1);
	}
}

int main(int argc, char **argv) {
	/*if(argc < 5){
	 std::cerr<<"Incorrect arguments. <particle count> <iterations> <save-image-every-x-iteration> <image-path>"<<std::endl;
	 exit(0);
	 }*/

	//printf("Size: %d\n", sizeof(Vec2f));
	//exit(0);

	std::cout<<"Running simulation with the following settings:\n";
	ConfigData config = parseConfigFile("nbodyConfig.txt");
	std::cout<<"=====================\n";
	//exit(0);

	const int particleCount = config.particleCount;//std::stoi(argv[1]);
	const int maxIteration = config.totalIterations;//std::stoi(argv[2]);
	const int imageEveryIteration = config.save_Image_Every_Xth_Iteration;//std::stoi(argv[3]);
	const float timestep = config.timestep;
	const float minBodyMass = config.minRandBodyMass;
	const float maxBodyMass = config.maxRandBodyMass;
	fieldWidth = config.fieldWidth;
	doubleFieldWidth = fieldWidth << 1;
	fieldHeight = config.fieldHeight;
	doubleFieldHeight = fieldHeight << 1;


	std::stringstream fileOutput;
	std::stringstream imgOut;
	//std::vector<Particle> bodies;
	//std::vector<float> updatedMasses;


//	std::mt19937 generator;
//	std::uniform_real_distribution<float> distribution(0.f, 1.f);
//	std::uniform_real_distribution<float> massDist(minBodyMass, maxBodyMass);
//	std::uniform_real_distribution<float> radiusDist(config.minRadius, config.maxRadius);
//	// distribution(generator);
//
//	//Randomly generating bodies
//	 for (int bodyIndex = 0; bodyIndex < particleCount; ++bodyIndex)
//	 {
//	 	x = (distribution(generator) * doubleFieldWidth) - fieldWidth; //gen.fval(0, doubleFieldWidth) - fieldWidth;
//	 	y = (distribution(generator) * doubleFieldHeight) - fieldHeight;
//	 	m = massDist(generator);
//	 	r = radiusDist(generator);//gen.fval(5, 50);
//	 	//printf("Generated Particle:\nPos: (%.4f,%.4f)\nMass: %.4f\nRadius: %.4f\n", x, y, m, r);
//	 	p = Particle(Vec2f(x,y), Vec2f(0.0,0.0), m, r);
//	 	bodies.push_back(p);
//	 	// updatedMasses.push_back(p.Mass);
//	 }


	//Particle p;
	float x, y, m, r;
	BodiesData bData;
	bData.alloc(particleCount);
	printf("Bodies: %d\n", bData.numBodies);

	jbutil::randgen gen;
	gen.seed(jbutil::gettime());

	//Randomly generating body data
	for (int bodyIndex = 0; bodyIndex < particleCount; ++bodyIndex)
	{
		 x = gen.fval(0, doubleFieldWidth) - fieldWidth;
		 y = gen.fval(0, doubleFieldHeight) - fieldHeight;
		 m = gen.fval(minBodyMass, maxBodyMass);
		 r = gen.fval(config.minRadius, config.maxRadius);
		 //printf("Base: %p\n Offset: %p\n", &bData.Positions, &bData.Velocities);
		 bData.Positions[bodyIndex] = Vec2f(x, y);
		 bData.Velocities[bodyIndex] = Vec2f(0.f, 0.f);
		 bData.Masses[bodyIndex] = m;
		 bData.Radii[bodyIndex] = r;
		 //p = Particle(Vec2f(x,y), Vec2f(0.0,0.0), m, r);
		 //bodies.push_back(p);
		 //updatedMasses.push_back(p.Mass);
	}
//	 bData.Positions[0] = Vec2f(-384400, 0);
//	 bData.Positions[1] = Vec2f(0.f, 0.f);
//	 bData.Velocities[0] = Vec2f(0.f, -37001.491f); //3701.491
//	 bData.Velocities[1] = Vec2f(0.f, 0.f);
//	 bData.Masses[0] = 7.35e22f;
//	 bData.Masses[1] = 5.97e24f;
//	 bData.Radii[0] = 1737.1f;
//	 bData.Radii[1] = 6371.0f;

//	 bData.Positions[0] = Vec2f(-500, 0);
//	 bData.Positions[1] = Vec2f(500.f, 0.f);
//	 bData.Positions[2] = Vec2f(-100.f, 550.f);
//	 bData.Velocities[0] = Vec2f(10.f, 0); //3701.491
//	 bData.Velocities[1] = Vec2f(-10.f, 0.f);
//	 bData.Velocities[2] = Vec2f(0.f, -8.f);
//	 bData.Masses[0] = 1e6f;
//	 bData.Masses[1] = 1e7f;
//	 bData.Masses[2] = 1e5f;
//	 bData.Radii[0] = 10.f;
//	 bData.Radii[1] = 20.f;
//	 bData.Radii[2] = 7.f;

	 //bData.printData();


	// updatedMasses.push_back(50.f);
	const int imgWidth = config.imgWidth;
	const int imgHeight = config.imgHeight;
	size_t imageSize = imgWidth * imgHeight;

	//printf("SIZE: %d\n", sizeof(Particle));
	float* d_updatedMasses;
	float* d_updatedRadii;
	Vec2f* d_updatedVelocities;
	//Particle* d_bodies;
	char* imgData;
	char* d_imgData;
	int threadsPerBlock = THREADS_PER_BLOCK;
	int blocks;

	BodiesData newData;
	int numBodies = particleCount;
	int newNumBodies;
	size_t sharedMemSize = threadsPerBlock * ((2 * (sizeof(Vec2f) + sizeof(float) + sizeof(float))) + 2 * sizeof(Vec2f));

	hipStream_t calculationStream;
	hipStream_t imageStream;

	hipStreamCreate(&calculationStream);
	hipStreamCreate(&imageStream);
	for (int run = 0; run < MAX_RUNS; ++run) {
		for (int iteration = 0; iteration < maxIteration; ++iteration)
		{
			//printf("Iteration: %d\n", iteration);
			//std::vector<Particle> newBodies;
			std::vector<float> updatedMasses(numBodies);
			std::vector<float> updatedRadii(numBodies);

			//printf("Bodies: %d\n", numBodies);
			//bData.printData();


			//hipMalloc((void **) &d_bodies, bodies.size() * sizeof(Particle));
			hipMalloc((void **) &d_updatedMasses, numBodies * sizeof(float));
			hipMalloc((void **) &d_updatedRadii, numBodies * sizeof(float));
			hipMalloc((void **) &d_updatedVelocities, numBodies * sizeof(Vec2f));



			for (int i = 0; i < numBodies; ++i) {
				updatedMasses[i] = bData.Masses[i];
				updatedRadii[i] = bData.Radii[i];
				//printf("Mass: %.4f \nRadius: %.4f\n", body.Mass, body.Radius);
			}



			//Since bodies can decrease, we need to ensure that at least 1 block is always present
			blocks = numBodies < threadsPerBlock ? 1 : numBodies / threadsPerBlock;
			//printf("BLOCKS: %d\n", blocks);

			//Copying data over to device
			bData.uploadToDevice(calculationStream);
//			bData.uploadToDevice();
			hipMemcpyAsync(d_updatedMasses, updatedMasses.data(), updatedMasses.size() * sizeof(float), hipMemcpyHostToDevice, calculationStream);
			hipMemcpyAsync(d_updatedRadii, updatedRadii.data(), updatedRadii.size() * sizeof(float), hipMemcpyHostToDevice, calculationStream);

//			hipMemcpy(d_updatedMasses, updatedMasses.data(), updatedMasses.size() * sizeof(float), hipMemcpyHostToDevice);
//			hipMemcpy(d_updatedRadii, updatedRadii.data(), updatedRadii.size() * sizeof(float), hipMemcpyHostToDevice);

			//Calculating movement
			ComputeForces<<<blocks, threadsPerBlock, sharedMemSize, calculationStream>>>
					(bData.d_contiguousData, d_updatedMasses, d_updatedVelocities, d_updatedRadii, numBodies,timestep, fieldWidth, fieldHeight, blocks);
			//CUDA_SYNC_CHECK();
			MoveBodies<<<blocks, threadsPerBlock, 0, calculationStream>>>(bData.d_contiguousData ,d_updatedMasses, d_updatedVelocities, d_updatedRadii, bData.numBodies, timestep);
			//CUDA_SYNC_CHECK();

			//DeleteMassesAndUpdateBodies<<<1,1>>>();

			//Copying data back to host
			hipMemcpyAsync(bData.contiguousData, bData.d_contiguousData, bData.size, hipMemcpyDeviceToHost, calculationStream);
//			hipMemcpy(bData.contiguousData, bData.d_contiguousData, bData.size, hipMemcpyDeviceToHost);
			//hipMemcpy(updatedMasses.data(), d_updatedMasses, updatedMasses.size() * sizeof(float), hipMemcpyDeviceToHost);

			newNumBodies = 0;
			for (size_t i = 0; i < numBodies; ++i) {
				if (bData.Masses[i] != 0.f)
				{
					newNumBodies++;
					//newBodies.push_back(bodies[i]);
				}
			}

			newData.alloc(newNumBodies);

			//copying relevant data to new place in memory while deleting other masses
			int newIdx = 0;
			for (size_t i = 0; i < numBodies; ++i)
			{
				if (bData.Masses[i] != 0.f)
				{
					 newData.Positions[newIdx] = bData.Positions[i];
					 newData.Velocities[newIdx] = bData.Velocities[i];
					 newData.Masses[newIdx] = bData.Masses[i];
					 newData.Radii[newIdx] = bData.Radii[i];
					 newIdx++;
				}
			}

			//Saving the image generated asynchronously in the previous iteration
			if ((iteration - 1) % imageEveryIteration == 0)
			{
				//printf("Hi\n");
				hipStreamSynchronize(imageStream);
				imgOut.str(std::string());
				imgOut << config.imagePath << "/iteration_" << iteration - 1 << ".ppm";
				//printf("Saving Iteration %d\n", iteration-1);
				saveImageToDisk(imgOut.str(), imgData, imgWidth, imgHeight);
				hipFree(d_imgData);
				delete[] imgData;
			}

			numBodies = newNumBodies;
			bData.freeData();
			bData = newData;

			if (iteration % imageEveryIteration == 0) {
				//printf("Hey\n");
				imgData = new char[imageSize];
				bData.uploadToDevice();
				hipMalloc((void**) &d_imgData, imageSize);
				hipMemsetAsync(d_imgData, 254, imageSize, imageStream);
//				hipMemset(d_imgData, 254, imageSize);
				generateImage<<<blocks, threadsPerBlock, threadsPerBlock * (sizeof(Vec2f) + sizeof(float)), imageStream>>>
						(bData.d_contiguousData, bData.numBodies, d_imgData, imgWidth, imgHeight, fieldWidth, fieldHeight);
				hipMemcpyAsync(imgData, d_imgData, imageSize, hipMemcpyDeviceToHost, imageStream);
//				hipMemcpy(imgData, d_imgData, imageSize, hipMemcpyDeviceToHost);

//				imgOut.str(std::string());
//				imgOut << config.imagePath << "/iteration_" << iteration<< ".ppm";
//				printf("Saving Iteration %d\n", iteration);
//				saveImageToDisk(imgOut.str(), imgData, imgWidth, imgHeight);
//				hipFree(d_imgData);
//				delete[] imgData;

			}
			hipFree(d_updatedMasses);
			hipFree(d_updatedRadii);
			hipFree(d_updatedVelocities);
			//printf("++++++++++++++++++++++++++++++++++++++++++++++++++++\n");
		}
			CUDA_SYNC_CHECK();
			hipDeviceReset();
	}
	return 0;
}
